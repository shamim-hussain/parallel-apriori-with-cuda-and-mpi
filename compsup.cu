#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "compsup.h"


__global__  void compute_support_kernel(char* patterns, size_t num_patterns, 
									char*  dataset, size_t num_data,
									size_t trans_len, unsigned int* supports){

	unsigned int ind_x=blockIdx.x * blockDim.x + threadIdx.x;
	char* pat_j = patterns+ind_x*trans_len;
	char* dat_i;
	char* dat_end = dataset+num_data*trans_len;

	if (ind_x>=num_patterns) return;

	unsigned int sup_j = 0;
	unsigned int k;
	int not_subset;
	for (dat_i = dataset; dat_i<dat_end;dat_i+=trans_len){
	not_subset=0;
	for (k=0;k<trans_len;k++){
	not_subset = not_subset | (pat_j[k]&(~dat_i[k]));
	}
	sup_j =sup_j + !not_subset;
	}
	supports[ind_x]=sup_j;
}


void Compute::set_data(char*  dataset, size_t num_data){
	g_num_data=num_data;
	if (g_dataset!=NULL) hipFree(g_dataset);
	hipMalloc(&g_dataset, g_num_data * g_trans_len * sizeof(char));
	hipMemcpy(g_dataset, dataset, g_num_data * g_trans_len * sizeof(char),
					hipMemcpyHostToDevice);
}

void Compute::allocate_data(size_t num_data){
	g_num_data=num_data;
	if (g_dataset!=NULL) hipFree(g_dataset);
	hipMallocManaged(&g_dataset, g_num_data * g_trans_len * sizeof(char));
}

char* Compute::get_data_addr(){
	return g_dataset;
}

void Compute::set_patterns(char* patterns, size_t num_patterns){
	g_num_patterns=num_patterns;
	if (g_patterns!=NULL)hipFree(g_patterns);				
	hipMalloc(&g_patterns, g_num_patterns * g_trans_len * sizeof(char)); 
	hipMemcpy(g_patterns, patterns, g_num_patterns * g_trans_len * sizeof(char),
				hipMemcpyHostToDevice);
}

void Compute::compute_support(){
		//threading info           
		size_t blocksCount = (g_num_patterns+threadsCount-1)/threadsCount;
		
		if (g_supports!=NULL) hipFree(g_supports);
		hipMallocManaged(&g_supports, g_num_patterns * sizeof(unsigned int));
		
		compute_support_kernel<<<blocksCount, threadsCount>>> (g_patterns, g_num_patterns,
																g_dataset, g_num_data, 
													 			g_trans_len, g_supports);
		
		hipDeviceSynchronize();
}


void Compute::get_supports(unsigned int* supports){
	hipMemcpy(supports, g_supports, g_num_patterns * sizeof(unsigned int),
				 hipMemcpyDeviceToHost);
}

void Compute::free_all(){
	if (g_dataset!=NULL) hipFree(g_dataset);
	if (g_patterns!=NULL)hipFree(g_patterns);
	if (g_supports!=NULL) hipFree(g_supports);
	g_dataset=NULL;
	g_patterns=NULL;
	g_supports=NULL;
}


// Function to initialize CUDA device
void cuda_init(int myrank)
{
    int cudaDeviceCount;
    hipError_t cE;
    
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
        printf(" Unable to determine cuda device count, error is %d, count is %d\n",
                                                                cE, cudaDeviceCount );
        exit(-1);
    }

    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess ){
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
        myrank, (myrank % cudaDeviceCount), cE);
        exit(-1);
    } 
}


