#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "compsup.h"

// Implemnts the Compute class and relevant functions for support computation



// Cuda kernel to compute support
__global__  void compute_support_kernel(char* patterns, size_t num_patterns, 
									char*  dataset, size_t num_data,
									size_t trans_len, unsigned int* supports){
	// Thread index
	unsigned int ind_x=blockIdx.x * blockDim.x + threadIdx.x;
	// Corresponding pattern
	char* pat_j = patterns+ind_x*trans_len;
	// datapoint pointer iterator
	char* dat_i;
	char* dat_end = dataset+num_data*trans_len;
	
	// We exceeded the array length
	if (ind_x>=num_patterns) return;
	
	// start calcuation of support
	unsigned int sup_j = 0;
	unsigned int k;
	int not_subset;
	
	for (dat_i = dataset; dat_i<dat_end;dat_i+=trans_len){
		not_subset=0;
		for (k=0;k<trans_len;k++){
			not_subset = not_subset | (pat_j[k]&(~dat_i[k]));
		}
		sup_j =sup_j + !not_subset;
	}

	// Accumulate supports
	supports[ind_x]+=sup_j;
}

// Copy over data to GpU memory
void Compute::set_data(char*  dataset, size_t num_data){
	g_num_data=num_data;
	if (g_dataset!=NULL) hipFree(g_dataset);
	hipMalloc(&g_dataset, g_num_data * g_trans_len * sizeof(char));
	hipMemcpy(g_dataset, dataset, g_num_data * g_trans_len * sizeof(char),
					hipMemcpyHostToDevice);
}

// Allocate data on GPU
void Compute::allocate_data(size_t num_data){
	g_num_data=num_data;
	if (g_dataset!=NULL) hipFree(g_dataset);
	hipMallocManaged(&g_dataset, g_num_data * g_trans_len * sizeof(char));
}


// Get the address of the data
char* Compute::get_data_addr(){
	return g_dataset;
}


// copy the patterns to GPU and allocate momory for support calcualtion
void Compute::set_patterns(char* patterns, size_t num_patterns){
	g_num_patterns=num_patterns;
	if (g_patterns!=NULL)hipFree(g_patterns);				
	hipMalloc(&g_patterns, g_num_patterns * g_trans_len * sizeof(char)); 
	hipMemcpy(g_patterns, patterns, g_num_patterns * g_trans_len * sizeof(char),
				hipMemcpyHostToDevice);

	if (g_supports!=NULL) hipFree(g_supports);
	hipMallocManaged(&g_supports, g_num_patterns * sizeof(unsigned int));
	hipMemset(g_supports,0,g_num_patterns * sizeof(unsigned int));
}


// Launches the kernel to comput the support
void Compute::compute_support(){
		//threading info           
		size_t blocksCount = (g_num_patterns+threadsCount-1)/threadsCount;
		
		compute_support_kernel<<<blocksCount, threadsCount>>> (g_patterns, g_num_patterns,
																g_dataset, g_num_data, 
													 			g_trans_len, g_supports);
		
		hipDeviceSynchronize();
}

// Copy back the computed supports
void Compute::get_supports(unsigned int* supports){
	hipMemcpy(supports, g_supports, g_num_patterns * sizeof(unsigned int),
				 hipMemcpyDeviceToHost);
}


// Free all allocated memories
void Compute::free_all(){
	if (g_dataset!=NULL) hipFree(g_dataset);
	if (g_patterns!=NULL)hipFree(g_patterns);
	if (g_supports!=NULL) hipFree(g_supports);
	g_dataset=NULL;
	g_patterns=NULL;
	g_supports=NULL;
}


// Function to initialize CUDA device
void cuda_init(int myrank)
{
    int cudaDeviceCount;
    hipError_t cE;
    
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
        printf(" Unable to determine cuda device count, error is %d, count is %d\n",
                                                                cE, cudaDeviceCount );
        exit(-1);
    }

    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess ){
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
        myrank, (myrank % cudaDeviceCount), cE);
        exit(-1);
    } 
}


