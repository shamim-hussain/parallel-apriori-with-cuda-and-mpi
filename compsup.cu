#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "compsup.h"

char* g_patterns=NULL;
char* g_dataset=NULL;
unsigned int* g_supports=NULL;

#define NUM_THREADS 256;


__global__  void compute_support_kernel(char* patterns, size_t num_patterns, 
										char*  dataset, size_t num_data,
										size_t trans_len, unsigned int* supports);

void hostToDeviceMemory(char* patterns, size_t num_patterns, 
						char*  dataset, size_t num_data,
						size_t trans_len, unsigned int* supports){
	
	//allocate data variables						
	hipMallocManaged(&g_patterns, num_patterns * trans_len * sizeof(char)); 
	hipMemcpy(g_patterns, patterns, num_patterns * trans_len * sizeof(char),
				hipMemcpyHostToDevice);
	
	hipMallocManaged(&g_dataset, num_data * trans_len * sizeof(char));
	hipMemcpy(g_dataset, dataset, num_data * trans_len * sizeof(char),
				 hipMemcpyHostToDevice);
	
	hipMallocManaged(&g_supports, num_patterns * sizeof(unsigned int));	
}

void compute_support(char* patterns, size_t num_patterns, 
						char*  dataset, size_t num_data, 
						size_t trans_len, unsigned int* supports){
		
		//threading info                    
		unsigned threadsCount = NUM_THREADS; // number of threads
		size_t blocksCount = (num_patterns+threadsCount-1)/threadsCount;

		//memory copy from host to device
		hostToDeviceMemory(patterns, num_patterns, dataset, num_data, trans_len, supports);

		//kernel invoke
		compute_support_kernel<<<blocksCount, threadsCount>>> (g_patterns, num_patterns,
																g_dataset, num_data, 
													 			trans_len, g_supports);
		
		hipDeviceSynchronize();
		hipMemcpy(supports, g_supports, num_patterns * sizeof(unsigned int),
				 hipMemcpyDeviceToHost);		
}

__global__  void compute_support_kernel(char* patterns, size_t num_patterns, 
						char*  dataset, size_t num_data,
						size_t trans_len, unsigned int* supports){
		
		unsigned int ind_x=blockIdx.x * blockDim.x + threadIdx.x;
		char* pat_j = patterns+ind_x*trans_len;
		char* dat_i;
		char* dat_end = dataset+num_data*trans_len;
		

		if (ind_x>=num_patterns) return;
		
		unsigned int sup_j = 0;
		unsigned int k;
		int not_subset;
		for (dat_i = dataset; dat_i<dat_end;dat_i+=trans_len){
			not_subset=0;
			for (k=0;k<trans_len;k++){
				not_subset = not_subset | (pat_j[k]&(~dat_i[k]));
			}
			sup_j =sup_j + !not_subset;
		}
		supports[ind_x]=sup_j;
}

// C version (for Neehal and Shoron)
void compute_support_n(char* patterns, size_t num_patterns, 
						char*  dataset, size_t num_data,
						size_t trans_len, unsigned int* supports)
{
	unsigned int sup_j, not_subset;
	char* pat_j;
	char* dat_i;

	size_t i,j,k;

	// Outer loop - iterates over patterns
	for (j=0,pat_j=patterns; j<num_patterns; j++, pat_j+=trans_len){
		sup_j=0;

		// Inner loop - iterates over transactions
		for (i=0,dat_i=dataset; i<num_data; i++, dat_i+=trans_len){
			not_subset=0;

			// Innermost loop - iterates over bytes
			for (k=0; k<trans_len; k++){
				not_subset = not_subset | (pat_j[k]&(~dat_i[k]));
			}

			sup_j = sup_j + !not_subset;
		}

		supports[j]=sup_j;
	}
}